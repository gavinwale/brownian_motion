#include "hip/hip_runtime.h"
/**
 * This program computes the brownian motion equation
 * in 1D using the CUDA interface for NVIDIA GPUs.
 * 
 * Written by: Gavin Wale
 *             ME471: Parallel Scientific Computing
 *             Boise State University
 *             5/1/2022
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include <hiprand/hiprand_kernel.h>

#define THREADS_PER_BLOCK 8 // to be adjusted up to 1024


__device__ float brownian(float sigma, int N, float dt);
__device__ float normal_dist();



__global__ void brownianOnGPU_globalAtomic(int n, float *dx_max, float *dx_min, float *dx_mean, float *dx_std) {

    float sigma = 0.1;
    int N = 100;
    float dt = 0.1;
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {

        float x = brownian(sigma, N, dt); // Every thread calculates x

        __syncthreads(); // Avoid data race
        if(x>*dx_max) *dx_max = x; // If x is > dx_max, new dx_max

        __syncthreads(); // Avoid data race
        if(x<*dx_min) *dx_min = x; // If x is < dx_min, new dx_min


        // atomicAdd(*dx_mean, x);
        __syncthreads();
        *dx_mean = *dx_mean + x;
        
        __syncthreads();
        
        float temp = x*x;
        
        *dx_std = *dx_std + temp;
        
        //atomicAdd(*dx_std,x*x);
        
    }
}

/**
 * Device function that can be called only by the GPU.
 * Returns the calculation of a brownian and makes an internal call
 * to another device function.
 * 
 * @param sigma 
 * @param N
 * @param dt 
 * @return x - a float representing brownian motion 
 */
__device__ float brownian(float sigma, int N, float dt) {
    float dW, mu;
    float x = 1; //initial value for the brownian motion experiment

    for (int i=1; i<N; i++){

        mu = sin(i*dt + M_PI/4); // mu
        dW = sqrt(dt)*normal_dist(); // noise
        x = x + mu*x*dt + sigma*x*dW;
    }
    return x;
}

/**
 * Device function to create noise for x.
 * Called in the device function "brownian".
 * 
 * @return x - a float with noise
 */
__device__ float normal_dist() {

    int id = threadIdx.x;
    hiprandState state;
    hiprand_init ( id + 1, id, 0, &state);
    float U1 = hiprand_uniform( &state);
    float U2 = hiprand_uniform( &state);

    float x = sqrt(-2*log(U1))*cos(2*M_PI*U2);
    return x;
    
}


/**
 * Main function, calls all necessary functions
 * to calculate the statistics of the brownian motion
 */
int main() {

    // Timer variables
    double iStart, iElaps;
    
    // Set up GPU
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    int N_experiments = 10000000; // 10^7 experiments

    // Allocate memory on host
    float *hx_max, *hx_min, *hx_mean, *hx_std;
    hx_max = (float *)malloc(sizeof(float));
    hx_min = (float *)malloc(sizeof(float));
    hx_mean = (float *)malloc(sizeof(float));
    hx_std = (float *)malloc(sizeof(float));
    //hx_max = 0;
    //hx_min = 1e7;
    //hx_mean = 0;
    //hx_std = 0; 

    // Allocate memoroy on device
    float *dx_max, *dx_min, *dx_mean, *dx_std;
    CHECK(hipMalloc((float **)&dx_max, sizeof(float)));
    CHECK(hipMalloc((float **)&dx_min, sizeof(float)));
    CHECK(hipMalloc((float **)&dx_mean, sizeof(float)));
    CHECK(hipMalloc((float **)&dx_std, sizeof(float)));

    // Set each value to what they should be on the device
    CHECK(hipMemset(dx_max, 0, sizeof(float)));
    CHECK(hipMemset(dx_min, 1e7, sizeof(float)));
    CHECK(hipMemset(dx_mean, 0, sizeof(float)));
    CHECK(hipMemset(dx_std, 0, sizeof(float)));

    //CHECK(hipMemcpy(dx_max, hx_max, sizeof(float), hipMemcpyHostToDevice));
    //CHECK(hipMemcpy(dx_min, hx_min, sizeof(float), hipMemcpyHostToDevice));
    //CHECK(hipMemcpy(dx_mean, hx_mean, sizeof(float), hipMemcpyHostToDevice));
    //CHECK(hipMemcpy(dx_std, hx_std, sizeof(float), hipMemcpyHostToDevice));

    // Number of threads and blocks to call below
    int nThreads = 8;
    int nBlocks = ((10000000 + 8 - 1) / 8);

    // Device runs brownian with input number of blocks and threads for N_experiments
    
    // Timer start
    iStart = seconds();
    
    brownianOnGPU_globalAtomic<<<nBlocks, nThreads>>>(N_experiments, dx_max, dx_min, dx_mean, dx_std);
    CHECK(hipDeviceSynchronize()); // No clue what this does

    // Print time it took to run brownianOnGPU_globalAtomic for specific threads, blocks
    // printf("brownianOnGPU_globalAtomic <<< %d, %d >>> Time elapsed: sec\n", nblocks, nThreads);

    // Check kernel error
    CHECK(hipGetLastError());

    // Send results from device to host
    CHECK(hipMemcpy(&hx_max, dx_max, sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(&hx_min, dx_min, sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(&hx_mean, dx_mean, sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(&hx_std, dx_std, sizeof(float), hipMemcpyDeviceToHost));
    
    // Timer stop
    iElaps = seconds() - iStart;

    // Calculating actual means and standard deviation off-device
    *hx_mean = *hx_mean / N_experiments;
    *hx_std = sqrt(*hx_std / N_experiments - *hx_mean * *hx_mean);

    // Print results to console on host
    printf("GPU time: %lf\n",iElaps);
    printf("Threads: %d   Blocks: %d\n",nThreads,nBlocks);
    printf("Global mean: %lf\n",hx_mean);
    printf("Global standard deviation: %lf\n",hx_std);
    printf("Global maximum: %lf\n",hx_max);
    printf("Global minimum: %lf\n",hx_min);

    // Free device global memory
    CHECK(hipFree(dx_max));
    CHECK(hipFree(dx_min));
    CHECK(hipFree(dx_mean));
    CHECK(hipFree(dx_std));

    // Free host memory
    free(hx_max);
    free(hx_min);
    free(hx_mean);
    free(hx_std);

    return(0);

}








